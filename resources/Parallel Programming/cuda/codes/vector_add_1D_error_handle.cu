
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

void my_cudasafe( hipError_t error, char* message)
{
	if(error!=hipSuccess) 
	{ 
		fprintf(stderr,"ERROR: %s : %s\n",message,hipGetErrorString(error)); 
		exit(-1); 
	}
}

__global__ void arradd(int* md, int* nd, int* pd, int size)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;
	
	pd[myid] = md[myid] + nd[myid];
}


int main()
{
	int size = 2000 * sizeof(int);
	int m[2000], n[2000], p[2000],*md, *nd,*pd;
	int i=0;

	
	for(i=0; i<2000; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}

	my_cudasafe(hipMalloc(&md, 1),"Cuda malloc : md");
	my_cudasafe(hipMemcpy(md, m, size, hipMemcpyHostToDevice),"Cuda memcopy H2D: md");

	my_cudasafe(hipMalloc(&nd, size),"Cuda malloc : nd");
	my_cudasafe(hipMemcpy(nd, n, size, hipMemcpyHostToDevice),"Cuda memcopy H2D: nd");

	my_cudasafe(hipMalloc(&pd, size),"Cuda malloc : pd");

	dim3   DimGrid(1000, 1);     
	dim3   DimBlock(20, 1);   


	arradd<<< DimGrid,DimBlock >>>(md,nd,pd,size);
	my_cudasafe(hipGetLastError(), "arradd kernel");

	my_cudasafe(hipMemcpy(p, pd, size, hipMemcpyDeviceToHost),"Cuda memcopy D2H: pd");
	my_cudasafe(hipFree(md),"hipFree md"); 
	my_cudasafe(hipFree(nd),"hipFree nd");
	my_cudasafe(hipFree(pd),"hipFree pd");

	for(i=0; i<2000; i++ )
	{
		printf("\t%d",p[i]);
	}	

}
