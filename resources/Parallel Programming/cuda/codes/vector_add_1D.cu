
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void arradd(int* md, int* nd, int* pd)
{
	int myid =  threadIdx.x;
	
	pd[myid] = md[myid] + nd[myid];
}


int main()
{
	int size = 200 * sizeof(int);
	int m[200], n[200], p[200],*md, *nd,*pd;
	int i=0;

	for(i=0; i<200; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}

	hipMalloc(&md, size);
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);

	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);

	hipMalloc(&pd, size);

	dim3   DimGrid(1, 1);     
	dim3   DimBlock(200, 1);   


	arradd<<< DimGrid,DimBlock >>>(md,nd,pd);

	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);

	for(i=0; i<200; i++ )
	{
		printf("\t%d",p[i]);
	}
		
	hipFree(md); 
	hipFree(nd);
	hipFree(pd);	
}

