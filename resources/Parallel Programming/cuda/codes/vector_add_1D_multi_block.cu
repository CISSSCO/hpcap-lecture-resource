
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
__global__ void arradd(int* md, int* nd, int* pd, int size)
{
	int myid = blockIdx.x*blockDim.x + threadIdx.x;	
	pd[myid] = md[myid] + nd[myid];
}
int main()
{
	int size = 2000 * sizeof(int);
	int m[2000], n[2000], p[2000],*md, *nd,*pd;
	int i=0;
	for(i=0; i<2000; i++ )
	{
		m[i] = i;
		n[i] = i;
		p[i] = 0;
	}
	hipMalloc(&md, size);
	hipMemcpy(md, m, size, hipMemcpyHostToDevice);
	hipMalloc(&nd, size);
	hipMemcpy(nd, n, size, hipMemcpyHostToDevice);
	hipMalloc(&pd, size);
	dim3   DimGrid(10, 1);     
	dim3   DimBlock(200, 1);   
	arradd<<< DimGrid,DimBlock >>>(md,nd,pd,size);
	hipMemcpy(p, pd, size, hipMemcpyDeviceToHost);
	hipFree(md); 
	hipFree(nd);
	hipFree (pd);

	for(i=0; i<2000; i++ )
	{
		printf("\t%d",p[i]);
	}	

}
